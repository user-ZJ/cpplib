#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <hipDNN.h>
#include "stdio.h"
#include <mpi.h>

using namespace std;

__global__ void VecAdd(float *A,float *B,float *C){
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void GetGPUInfo(){
    int device_count;
    hipGetDeviceCount(&device_count); // GPU个数
    for (int i = 0; i < device_count; ++i)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device name: " << prop.name << std::endl;
        std::cout << "totalGlobalMem: " << prop.totalGlobalMem / 1024.0 / 1024 << "MB" << std::endl;
        // computeMode：设备计算模式。
        // computeCapabilityMajor和computeCapabilityMinor：设备的计算能力版本号。
        size_t free_byte, total_byte;
        hipMemGetInfo(&free_byte, &total_byte);
        std::cout << "Total memory: " << total_byte / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "Free memory: " << free_byte / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "used memory: " << (total_byte-free_byte) / (1024.0 * 1024.0) << " MB" << std::endl;
    }
}


int main()
{
    MPI_Init(NULL, NULL);
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the process
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    std::cout<<"word_size:"<<world_size<<" world_rank:"<<world_rank<<"\n";

    if(world_rank==0)
        GetGPUInfo();
    

    float *A,*B,*C;
    hipMalloc(&A, sizeof(float)*10);
    hipMalloc(&B, sizeof(float)*10);
    hipMalloc(&C, sizeof(float)*10);
    VecAdd<<<1,10>>>(A,B,C);
    MPI_Finalize();
    return 0;
}