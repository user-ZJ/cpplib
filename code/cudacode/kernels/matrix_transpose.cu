
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

// #define N 2048
#define BLOCK_SIZE 32 

__global__ void matrix_transpose_naive(float *input, float *output,int N) {

	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;
	int index = indexY * N + indexX;
	int transposedIndex = indexX * N + indexY;

	// this has discoalesced global memory store  
	output[transposedIndex] = input[index];

	// this has discoalesced global memore load
	// output[index] = input[transposedIndex];
}

__global__ void matrix_transpose_shared(float *input, float *output,int N) {
    // 使用BLOCK_SIZE + 1，解决wrap的bank冲突
	__shared__ int sharedMemory [BLOCK_SIZE] [BLOCK_SIZE + 1];

	// global index	
	int indexX = threadIdx.x + blockIdx.x * blockDim.x;
	int indexY = threadIdx.y + blockIdx.y * blockDim.y;

	// transposed global memory index
	int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
	int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

	// local index
	int localIndexX = threadIdx.x;
	int localIndexY = threadIdx.y;

	int index = indexY * N + indexX;
	int transposedIndex = tindexY * N + tindexX;

	// reading from global memory in coalesed manner and performing tanspose in shared memory
	sharedMemory[localIndexX][localIndexY] = input[index];

	__syncthreads();

	// writing into global memory in coalesed fashion via transposed data in shared memory
	output[transposedIndex] = sharedMemory[localIndexY][localIndexX];
}


int matrix_transpose(float *d_out, float *d_in,int N) {
	dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 gridSize(N/BLOCK_SIZE,N/BLOCK_SIZE,1);

	matrix_transpose_shared<<<gridSize,blockSize>>>(d_in,d_out,N);
	return 0;
}

int global_matrix_transpose(float *d_out, float *d_in,int N) {
	dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
	dim3 gridSize(N/BLOCK_SIZE,N/BLOCK_SIZE,1);

	matrix_transpose_naive<<<gridSize,blockSize>>>(d_in,d_out,N);
	return 0;
}





