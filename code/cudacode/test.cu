#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <cudnn.h>
#include "stdio.h"

using namespace std;

__global__ void VecAdd(float *A,float *B,float *C){
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}


int main()
{
    int device_count;
    hipGetDeviceCount(&device_count); // GPU个数
    for (int i = 0; i < device_count; ++i)
    {
        hipDeviceProp_t prop;
        hipError_t error = hipGetDeviceProperties(&prop, i);
        std::cout << "Device name: " << prop.name << std::endl;
        std::cout << "totalGlobalMem: " << prop.totalGlobalMem / 1024.0 / 1024 << "MB" << std::endl;
        // computeMode：设备计算模式。
        // computeCapabilityMajor和computeCapabilityMinor：设备的计算能力版本号。
        size_t free_byte, total_byte;
        hipMemGetInfo(&free_byte, &total_byte);
        std::cout << "Total memory: " << total_byte / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "Free memory: " << free_byte / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "used memory: " << (total_byte-free_byte) / (1024.0 * 1024.0) << " MB" << std::endl;
    }

    float *A,*B,*C;
    hipMalloc(&A, sizeof(float)*10);
    hipMalloc(&B, sizeof(float)*10);
    hipMalloc(&C, sizeof(float)*10);
    VecAdd<<<1,10>>>(A,B,C);

    return 0;
}